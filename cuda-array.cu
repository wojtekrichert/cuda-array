
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <ctime>
#include <stdlib.h>
using namespace std;
const int BLOCK_SIZE = 128;
const long int size = 134217728;


double  dodawanie(double *array){
  double suma =0;
  for(int i =0; i <size; i++){
    suma = suma + array[i];
  }
  return suma;
}

__global__  void total(double * input, double * output, long int len)
{
    __shared__ double suma_czesciowa[2*BLOCK_SIZE];
    int globalThreadId = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int t = threadIdx.x;
    unsigned int start = 2*blockIdx.x*blockDim.x;

    if ((start + t) < len)
    {
        suma_czesciowa[t] = input[start + t];
    }
    else
    {
        suma_czesciowa[t] = 0.0;
    }
    if ((start + blockDim.x + t) < len)
    {
        suma_czesciowa[blockDim.x + t] = input[start + blockDim.x + t];
    }
    else
    {
        suma_czesciowa[blockDim.x + t] = 0.0;
    }

    for (unsigned long long q = blockDim.x; q > 0; q /= 2)
    {
      __syncthreads();
        if (t < q)
            suma_czesciowa[t] += suma_czesciowa[t + q];
    }
    __syncthreads();

    if (t == 0 && (globalThreadId*2) < len)
    {
        output[blockIdx.x] = suma_czesciowa[t];
    }
}


int main(void){
  double *array= new double[size];
  for(int i =0; i<size;i++){
    array[i]=(rand() % 100 + 1)*(rand() % 100 + 1);
  }
  //Kod na CPU:--------------------------------------------
  clock_t begin = clock();
  double suma =dodawanie(array);
  printf("%f ", suma);
  clock_t end = clock();
  double elapsed_secs = (double(end - begin) / CLOCKS_PER_SEC);
  printf("czas CPU: %f \n", elapsed_secs);
  //-------------------------------------------------------
  double * d_array;
  hipMalloc(&d_array, size*sizeof(double));


  double * arrayOUT;
  double * d_arrayOUT;
  double numOutputElements;
  numOutputElements = size / (BLOCK_SIZE<<1);
  if (size % (BLOCK_SIZE<<1))
  {
      numOutputElements++;
  }
  arrayOUT = (double*) malloc(numOutputElements * sizeof(double));

  hipMalloc((void **)&d_arrayOUT, numOutputElements * sizeof(double));
  hipMemcpy(d_array, array, size * sizeof(double), hipMemcpyHostToDevice);

  dim3 DimGrid( numOutputElements, 1, 1);
  dim3 DimBlock(BLOCK_SIZE, 1, 1);

  clock_t begin1 = clock();
  total<<<DimGrid, DimBlock>>>(d_array, d_arrayOUT, size);
  hipMemcpy(arrayOUT, d_arrayOUT, numOutputElements * sizeof(double), hipMemcpyDeviceToHost);
  for (int ii = 1; ii < numOutputElements; ii++)
  {
      arrayOUT[0] += arrayOUT[ii];
  }
  clock_t end1 = clock();
  printf("%f ", arrayOUT[0]);
  double elapsed_secs1 = (double(end1 - begin1) / CLOCKS_PER_SEC);
  printf("czas CPU: %f \n", elapsed_secs1);
  printf("przyspieszenie: %f razy\n", (elapsed_secs/elapsed_secs1));

  // Free the GPU memory here
  hipFree(d_array);
  hipFree(d_arrayOUT);
  free(arrayOUT);



free(array);
}
