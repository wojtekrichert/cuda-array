
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <ctime>
#include <stdlib.h>
using namespace std;
const unsigned long long int size = 2147483648;

unsigned long long int  dodawanie(unsigned long long int *array){
  unsigned long long int suma =0;
  for(unsigned long long int i =0; i <size; i++){
    suma = suma + array[i];
  }
  printf("%llu ", suma);
  return suma;
}

int main(void){
  unsigned long long int *array= new unsigned long long int[size];
  for(unsigned long long int i =0; i<size;i++){
    array[i]=rand() % 100 + 1;
  }
  //Kod na CPU:--------------------------------------------
  clock_t begin = clock();
  unsigned long long int suma =dodawanie(array);
  printf("%llu ", suma);
  clock_t end = clock();
  double elapsed_secs = (double(end - begin) / CLOCKS_PER_SEC);
  printf("czas CPU: %f \n", elapsed_secs);
  //-------------------------------------------------------



free(array);
}
